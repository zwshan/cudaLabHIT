#include <stdio.h>
#include <hip/hip_runtime.h>
 
void printDeviceProp(const hipDeviceProp_t &prop)
{
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %zu.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %zu.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %zu.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %zu.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %zu.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}
 
bool InitCUDA()
{
    //used to count the device numbers
    int count;
 
    // get the cuda device count
    hipGetDeviceCount(&count);
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }
 
    // find the device >= 1.X
    int i;
    for (i = 0; i < count; ++i) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                printDeviceProp(prop);
                break;
            }
        }
    }
 
    // if can't find the device
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }
 
    // set cuda device
    hipSetDevice(i);
 
    return true;
}
 
int main(int argc, char const *argv[])
{
    if (InitCUDA()) {
        printf("CUDA initialized.\n");
    }
 
    return 0;
}